#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include <vector>
#include <memory>
#include <iostream>
using std::cerr;
using std::cout;
using std::endl;
#include <sstream>
#include <iomanip>
#include <string>
#include <fstream>
#include <omp.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using thrust::device_vector;
using thrust::host_vector;
#include <thrust/version.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/gather.h>
#include <typeinfo>

#include "hd/pipeline.h"
#include "hd/maths.h"
#include "hd/clean_filterbank_rfi.h"

#include "hd/remove_baseline.h"
#include "hd/matched_filter.h"
#include "hd/get_rms.h"
#include "hd/find_giants.h"
#include "hd/label_candidate_clusters.h"
#include "hd/merge_candidates.h"

#include "hd/DataSource.h"
#include "hd/ClientSocket.h"
#include "hd/SocketException.h"
#include "hd/stopwatch.h" // For benchmarking
// #include "hd/write_time_series.h" // For debugging

#include <dedisp.h>
#define CHECK(call)                               \
  do                                              \
  {                                               \
    const hipError_t error_code = call;          \
    if (error_code != hipSuccess)                \
    {                                             \
      printf("CUDA Error:\n");                    \
      printf("    File:       %s\n", __FILE__);   \
      printf("    Line:       %d\n", __LINE__);   \
      printf("    Error code: %d\n", error_code); \
      printf("    Error text: %s\n",              \
             hipGetErrorString(error_code));     \
      exit(1);                                    \
    }                                             \
  } while (0)

#define HD_BENCHMARK

#ifdef HD_BENCHMARK
void start_timer(Stopwatch &timer) { timer.start(); }
void stop_timer(Stopwatch &timer)
{
  hipDeviceSynchronize();
  timer.stop();
}
#else
void start_timer(Stopwatch &timer) {}
void stop_timer(Stopwatch &timer) {}
#endif // HD_BENCHMARK

#include <utility> // For std::pair
template <typename T, typename U>
std::pair<T &, U &> tie(T &a, U &b) { return std::pair<T &, U &>(a, b); }

const int N_threads = 7; // 设置流的数目
#define THRUST_DEBUG 1

struct hd_pipeline_t
{
  hd_params params;
  dedisp_plan dedispersion_plan;
  // MPI_Comm    communicator;
  // const int N_threads = 1;

  // Memory buffers used during pipeline execution
  std::vector<hd_byte> h_clean_filterbank;
  host_vector<hd_byte> h_dm_series;

  std::vector<device_vector<hd_float>> d_time_series;
  std::vector<device_vector<hd_float>> d_filtered_series;
  // device_vector<hd_float> d_time_series;
  // device_vector<hd_float> d_filtered_series;
  // hd_float *d_time_series;
  // hd_float *d_filtered_series;
};

hd_error allocate_gpu(const hd_pipeline pl)
{
  // TODO: This is just a simple proc-->GPU heuristic to get us started
  int gpu_count;
  hipGetDeviceCount(&gpu_count);
  // int proc_idx;
  // MPI_Comm comm = pl->communicator;
  // MPI_Comm_rank(comm, &proc_idx);
  int proc_idx = pl->params.beam;
  int gpu_idx = pl->params.gpu_id;

  hipError_t cerror = hipSetDevice(gpu_idx);
  if (cerror != hipSuccess)
  {
    cerr << "Could not setCudaDevice to " << gpu_idx << ": " << hipGetErrorString(cerror) << endl;
    return throw_cuda_error(cerror);
  }

  if (pl->params.verbosity >= 1)
  {
    cout << "Process " << proc_idx << " using GPU " << gpu_idx << endl;
  }

  if (!pl->params.yield_cpu)
  {
    if (pl->params.verbosity >= 2)
    {
      cout << "\tProcess " << proc_idx << " setting CPU to spin" << endl;
    }
    cerror = hipSetDeviceFlags(hipDeviceScheduleSpin);
    if (cerror != hipSuccess)
    {
      return throw_cuda_error(cerror);
    }
  }
  else
  {
    if (pl->params.verbosity >= 2)
    {
      cout << "\tProcess " << proc_idx << " setting CPU to yield" << endl;
    }
    // Note: This Yield flag doesn't seem to work properly.
    //   The BlockingSync flag does the job, although it may interfere
    //     with GPU/CPU overlapping (not currently used).
    // cerror = hipSetDeviceFlags(hipDeviceScheduleYield);
    cerror = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if (cerror != hipSuccess)
    {
      return throw_cuda_error(cerror);
    }
  }

  return HD_NO_ERROR;
}

unsigned int get_filter_index(unsigned int filter_width)
{
  // This function finds log2 of the 32-bit power-of-two number v
  unsigned int v = filter_width;
  static const unsigned int b[] = {0xAAAAAAAA, 0xCCCCCCCC, 0xF0F0F0F0,
                                   0xFF00FF00, 0xFFFF0000};
  register unsigned int r = (v & b[0]) != 0;
  for (int i = 4; i > 0; --i)
  {
    r |= ((v & b[i]) != 0) << i;
  }
  return r;
}
// 检查 CUDA 错误的辅助函数
void checkThrust(const char *context)
{
  hipError_t err = hipGetLastError();

  // 先同步设备，确保没有未决的错误
  hipDeviceSynchronize();

  if (err != hipSuccess)
  {
    std::cerr << "Error before " << context << ": " << hipGetErrorString(err) << std::endl;
  }
}

hd_error hd_create_pipeline(hd_pipeline *pipeline_, hd_params params)
{
  *pipeline_ = 0;

  // Note: We use a smart pointer here to automatically clean up after errors
#if __cplusplus <= 199711L
  typedef std::auto_ptr<hd_pipeline_t> smart_pipeline_ptr;
#else
  typedef std::unique_ptr<hd_pipeline_t> smart_pipeline_ptr;
#endif
  smart_pipeline_ptr pipeline = smart_pipeline_ptr(new hd_pipeline_t());
  if (!pipeline.get())
  {
    return throw_error(HD_MEM_ALLOC_FAILED);
  }

  pipeline->params = params;

  if (params.verbosity >= 2)
  {
    cout << "\tAllocating GPU..." << endl;
  }

  hd_error error = allocate_gpu(pipeline.get());
  if (error != HD_NO_ERROR)
  {
    return throw_error(error);
  }

  if (params.verbosity >= 3)
  {
    cout << "nchans = " << params.nchans << endl;
    cout << "dt     = " << params.dt << endl;
    cout << "f0     = " << params.f0 << endl;
    cout << "df     = " << params.df << endl;
  }

  if (params.verbosity >= 2)
  {
    cout << "\tCreating dedispersion plan..." << endl;
  }

  dedisp_error derror;
  derror = dedisp_create_plan(&pipeline->dedispersion_plan,
                              params.nchans, params.dt,
                              params.f0, params.df);
  if (derror != DEDISP_NO_ERROR)
  {
    return throw_dedisp_error(derror);
  }
  // TODO: Consider loading a pre-generated DM list instead for flexibility
  derror = dedisp_generate_dm_list(pipeline->dedispersion_plan,
                                   pipeline->params.dm_min,
                                   pipeline->params.dm_max,
                                   pipeline->params.dm_pulse_width,
                                   pipeline->params.dm_tol);
  if (derror != DEDISP_NO_ERROR)
  {
    return throw_dedisp_error(derror);
  }

  if (pipeline->params.use_scrunching)
  {
    derror = dedisp_enable_adaptive_dt(pipeline->dedispersion_plan,
                                       pipeline->params.dm_pulse_width,
                                       pipeline->params.scrunch_tol);
    if (derror != DEDISP_NO_ERROR)
    {
      return throw_dedisp_error(derror);
    }
  }

  *pipeline_ = pipeline.release();

  if (params.verbosity >= 2)
  {
    cout << "\tInitialisation complete." << endl;
  }

  if (params.verbosity >= 1)
  {
    cout << "Using Thrust v"
         << THRUST_MAJOR_VERSION << "."
         << THRUST_MINOR_VERSION << "."
         << THRUST_SUBMINOR_VERSION << endl;
  }

  return HD_NO_ERROR;
}

hd_error hd_execute(hd_pipeline pl,
                    const hd_byte *h_filterbank, hd_size nsamps, hd_size nbits,
                    hd_size first_idx, hd_size *nsamps_processed)
{
  hd_error error = HD_NO_ERROR;

  Stopwatch total_timer;
  Stopwatch memory_timer;
  Stopwatch clean_timer;
  Stopwatch dedisp_timer;
  Stopwatch communicate_timer;
  Stopwatch copy_timer;
  Stopwatch baseline_timer;
  Stopwatch normalise_timer;
  Stopwatch filter_timer;
  Stopwatch coinc_timer;
  Stopwatch giants_timer;
  Stopwatch candidates_timer;

  start_timer(total_timer);

  start_timer(clean_timer);
  // Note: Filterbank cleaning must be done out-of-place
  hd_size nbytes = nsamps * pl->params.nchans * nbits / 8;
  start_timer(memory_timer);
  pl->h_clean_filterbank.resize(nbytes);
  std::vector<int> h_killmask(pl->params.nchans, 1);
  stop_timer(memory_timer);

  if (pl->params.verbosity >= 2)
  {
    cout << "\tCleaning 0-DM filterbank..." << endl;
  }

  // Start by cleaning up the filterbank based on the zero-DM time series
  hd_float cleaning_dm = 0.f;
  if (pl->params.verbosity >= 3)
  {
    /*
    cout << "\tWriting dirty filterbank to disk..." << endl;
    write_host_filterbank(&h_filterbank[0],
                          pl->params.nchans, nsamps, nbits,
                          pl->params.dt, pl->params.f0, pl->params.df,
                          "dirty_filterbank.fil");
    */
  }
  // Note: We only clean the narrowest zero-DM signals; otherwise we
  //         start removing real stuff from higher DMs.
  error = clean_filterbank_rfi(pl->dedispersion_plan,
                               &h_filterbank[0],
                               nsamps,
                               nbits,
                               &pl->h_clean_filterbank[0],
                               &h_killmask[0],
                               cleaning_dm,
                               pl->params.dt,
                               pl->params.baseline_length,
                               pl->params.rfi_tol,
                               pl->params.rfi_min_beams,
                               pl->params.rfi_broad,
                               pl->params.rfi_narrow,
                               1); // pl->params.boxcar_max);
  if (error != HD_NO_ERROR)
  {
    return throw_error(error);
  }

  if (pl->params.verbosity >= 2)
  {
    cout << "Applying manual killmasks" << endl;
  }

  error = apply_manual_killmasks(pl->dedispersion_plan,
                                 &h_killmask[0],
                                 pl->params.num_channel_zaps,
                                 pl->params.channel_zaps);
  if (error != HD_NO_ERROR)
  {
    return throw_error(error);
  }

  hd_size good_chan_count = thrust::reduce(h_killmask.begin(),
                                           h_killmask.end());
  hd_size bad_chan_count = pl->params.nchans - good_chan_count;
  if (pl->params.verbosity >= 2)
  {
    cout << "Bad channel count = " << bad_chan_count << endl;
  }

  // TESTING
  // h_clean_filterbank.assign(h_filterbank, h_filterbank+nbytes);

  stop_timer(clean_timer);

  if (pl->params.verbosity >= 3)
  {
    /*
    cout << "\tWriting killmask to disk..." << endl;
    std::ofstream killfile("killmask.dat");
    for( size_t i=0; i<h_killmask.size(); ++i ) {
      killfile << h_killmask[i] << "\n";
    }
    killfile.close();

    cout << "\tWriting cleaned filterbank to disk..." << endl;
    write_host_filterbank(&pl->h_clean_filterbank[0],
                          pl->params.nchans, nsamps, nbits,
                          pl->params.dt, pl->params.f0, pl->params.df,
                          "clean_filterbank.fil");
    */
  }
  if (pl->params.verbosity >= 2)
  {
    cout << "\tGenerating DM list..." << endl;
  }

  if (pl->params.verbosity >= 3)
  {
    cout << "dm_min = " << pl->params.dm_min << endl;
    cout << "dm_max = " << pl->params.dm_max << endl;
    cout << "dm_tol = " << pl->params.dm_tol << endl;
    cout << "dm_pulse_width = " << pl->params.dm_pulse_width << endl;
    cout << "nchans = " << pl->params.nchans << endl;
    cout << "dt = " << pl->params.dt << endl;

    cout << "dedisp nchans = " << dedisp_get_channel_count(pl->dedispersion_plan) << endl;
    cout << "dedisp dt = " << dedisp_get_dt(pl->dedispersion_plan) << endl;
    cout << "dedisp f0 = " << dedisp_get_f0(pl->dedispersion_plan) << endl;
    cout << "dedisp df = " << dedisp_get_df(pl->dedispersion_plan) << endl;
  }

  hd_size dm_count = dedisp_get_dm_count(pl->dedispersion_plan);
  const float *dm_list = dedisp_get_dm_list(pl->dedispersion_plan);

  const dedisp_size *scrunch_factors =
      dedisp_get_dt_factors(pl->dedispersion_plan);
  if (pl->params.verbosity >= 3)
  {
    cout << "DM List for " << pl->params.dm_min << " to " << pl->params.dm_max << endl;
    for (hd_size i = 0; i < dm_count; ++i)
    {
      cout << dm_list[i] << endl;
    }
  }

  if (pl->params.verbosity >= 2)
  {
    cout << "Scrunch factors:" << endl;
    for (hd_size i = 0; i < dm_count; ++i)
    {
      cout << scrunch_factors[i] << " ";
    }
    cout << endl;
  }

  // Set channel killmask for dedispersion
  dedisp_set_killmask(pl->dedispersion_plan, &h_killmask[0]);
  if (dedisp_get_max_delay(pl->dedispersion_plan) > nsamps)
  {
    cerr << "maximum DM delay=" << dedisp_get_max_delay(pl->dedispersion_plan) << endl;
    cerr << "Number of samples=" << nsamps << endl;
    return throw_error(HD_TOO_FEW_NSAMPS);
  }

  hd_size nsamps_computed = nsamps - dedisp_get_max_delay(pl->dedispersion_plan);
  hd_size series_stride = nsamps_computed;

  // Report the number of samples that will be properly processed
  *nsamps_processed = nsamps_computed - pl->params.boxcar_max;

  if (pl->params.verbosity >= 3)
  {
    cout << "dm_count = " << dm_count << endl;
    cout << "max delay = " << dedisp_get_max_delay(pl->dedispersion_plan) << endl;
    cout << "nsamps_computed = " << nsamps_computed << endl;
  }

  hd_size beam = pl->params.beam;

  if (pl->params.verbosity >= 2)
  {
    cout << "\tAllocating memory for pipeline computations..." << endl;
  }

  start_timer(memory_timer);
  // 分配缓冲区内存
  pl->d_time_series.resize(N_threads);
  pl->d_filtered_series.resize(N_threads);
  pl->h_dm_series.resize(series_stride * pl->params.dm_nbits / 8 * dm_count);
  for (int i = 0; i < N_threads; ++i)
  {
    pl->d_time_series[i].resize(series_stride);
    pl->d_filtered_series[i].resize(series_stride, 0);
  }

  // hipMalloc(&d_time_series, series_stride * sizeof(hd_float) * N_threads);
  // hipMalloc(&d_filtered_series, series_stride * sizeof(hd_float) * N_threads);

  stop_timer(memory_timer);

  RemoveBaselinePlan baseline_remover;
  GetRMSPlan rms_getter;
  MatchedFilterPlan<hd_float> matched_filter_plan;
  GiantFinder giant_finder;
  std::vector<thrust::device_vector<hd_float>> d_giant_peaks_t(N_threads);
  std::vector<thrust::device_vector<hd_size>> d_giant_inds_t(N_threads);
  std::vector<thrust::device_vector<hd_size>> d_giant_begins_t(N_threads);
  std::vector<thrust::device_vector<hd_size>> d_giant_ends_t(N_threads);

  std::vector<thrust::device_vector<hd_size>> d_giant_filter_inds_t(N_threads);
  std::vector<thrust::device_vector<hd_size>> d_giant_dm_inds_t(N_threads);
  std::vector<thrust::device_vector<hd_size>> d_giant_members_t(N_threads);
  // thrust::device_vector<hd_float> d_giant_peaks;
  // thrust::device_vector<hd_size> d_giant_inds;
  // thrust::device_vector<hd_size> d_giant_begins;
  // thrust::device_vector<hd_size> d_giant_ends;
  // thrust::device_vector<hd_size> d_giant_filter_inds;
  // thrust::device_vector<hd_size> d_giant_dm_inds;
  // thrust::device_vector<hd_size> d_giant_members;

  typedef thrust::device_ptr<hd_float> dev_float_ptr;
  typedef thrust::device_ptr<hd_size> dev_size_ptr;

  if (pl->params.verbosity >= 2)
  {
    cout << "\tDedispersing for DMs " << dm_list[0]
         << " to " << dm_list[dm_count - 1] << "..." << endl;
  }

  // Dedisperse
  dedisp_error derror;
  const dedisp_byte *in = &pl->h_clean_filterbank[0];
  dedisp_byte *out = &pl->h_dm_series[0];
  dedisp_size in_nbits = nbits;
  dedisp_size in_stride = pl->params.nchans * in_nbits / 8;
  dedisp_size out_nbits = pl->params.dm_nbits;
  dedisp_size out_stride = series_stride * out_nbits / 8;
  unsigned flags = 0;
  start_timer(dedisp_timer);
  derror = dedisp_execute_adv(pl->dedispersion_plan, nsamps,
                              in, in_nbits, in_stride,
                              out, out_nbits, out_stride,
                              flags);
  // 执行去分散操作，将输入的滤波信号转换为去分散后的信号
  // dedisp_error derror = dedisp_execute_adv(pl->dedispersion_plan, nsamps,
  //                                          pl->h_clean_filterbank.data(), // 输入滤波后的原始信号
  //                                          nbits,                         // 输入信号的每个样本位宽
  //                                          pl->params.nchans * nbits / 8, // 输入信号的跨度（每个时间点的总字节数）
  //                                          pl->h_dm_series.data(),        // 输出去分散信号存储位置
  //                                          pl->params.dm_nbits,           // 输出信号的每个样本位宽
  //                                          series_stride * pl->params.dm_nbits / 8, // 输出信号的跨度
  //                                          0);                            // 执行参数标志
  stop_timer(dedisp_timer);
  if (derror != DEDISP_NO_ERROR)
  {
    return throw_dedisp_error(derror);
  }

  if (beam == 0 && first_idx == 0)
  {
    // TESTING
    // write_host_time_series((unsigned int*)out, nsamps_computed, out_nbits,
    //                       pl->params.dt, "dedispersed_0.tim");
  }

  if (pl->params.verbosity >= 2)
  {
    cout << "\tBeginning inner pipeline..." << endl;
  }

  // TESTING
  // hd_size write_dm = 0;

  bool too_many_giants = false;

  // 锁页内存分配
  dedisp_byte *pinned_dm_series;
  CHECK(hipHostAlloc(&pinned_dm_series, pl->h_dm_series.size() * sizeof(hd_byte), hipHostMallocDefault));
  printf("h_dm_series.size = %d, type:%s\n", pl->h_dm_series.size(), typeid(hd_byte).name());

  // 将原始数据移入锁页内存
  std::memcpy(pinned_dm_series, thrust::raw_pointer_cast(pl->h_dm_series.data()), pl->h_dm_series.size() * sizeof(hd_byte));

  // 配置 CUDA 流数量和 OpenMP 线程数量

  std::vector<hipStream_t> streams(N_threads);
  for (int i = 0; i < N_threads; ++i)
  {
    CHECK(hipStreamCreate(&streams[i]));
  }
  // #pragma omp parallel
  omp_set_num_threads(N_threads);
#pragma omp parallel
  {

    unsigned int num_threads = omp_get_num_threads();
    int tid = omp_get_thread_num();
    // CHECK(hipSetDevice(tid));
    // printf("Thread %d starting\n", tid);
    printf("Thread %d starting of %d\n", tid, num_threads);
    hipStream_t stream = streams[tid];
    hd_size chunk_size = dm_count / N_threads;
    hd_size start_idx = tid * chunk_size;
    hd_size end_idx = (tid == N_threads - 1) ? dm_count : (tid + 1) * chunk_size; // 最后一个线程特殊处理

    for (hd_size dm_idx = start_idx; dm_idx < end_idx; ++dm_idx)
    {
      hd_size cur_dm_scrunch = scrunch_factors[dm_idx];
      hd_size cur_nsamps = nsamps_computed / cur_dm_scrunch;
      hd_float cur_dt = pl->params.dt * cur_dm_scrunch;

      if (pl->params.verbosity >= 3)
      {
        cout << "dm_idx     = " << dm_idx << endl;
        cout << "scrunch    = " << scrunch_factors[dm_idx] << endl;
        cout << "cur_nsamps = " << cur_nsamps << endl;
        cout << "dt0        = " << pl->params.dt << endl;
        cout << "cur_dt     = " << cur_dt << endl;
        cout << "\tBaselining and normalising each beam..." << endl;
      }
      // 每个线程使用不同的缓冲区位置
      checkThrust("thrust::transform 582\n");
      hd_float *time_series = thrust::raw_pointer_cast(pl->d_time_series[tid].data());
      checkThrust("thrust::transform 582\n");
      printf("dm_idx = %d, tid = %d\n", dm_idx, tid);

      // 偏移量计算
      hd_size offset = dm_idx * series_stride * pl->params.dm_nbits / 8;

      // 根据位宽将锁页内存数据传输到设备并将数据转换为float类型
      switch (pl->params.dm_nbits)
      {
      case 8:
        CHECK(hipMemcpyAsync(time_series,
                              &pinned_dm_series[offset],
                              cur_nsamps * sizeof(dedisp_byte),
                              hipMemcpyHostToDevice, stream));
        break;
      case 16:
        CHECK(hipMemcpyAsync(time_series,
                              reinterpret_cast<unsigned short *>(&pinned_dm_series[offset]),
                              cur_nsamps * sizeof(unsigned short),
                              hipMemcpyHostToDevice, stream));
        break;
      case 32:
        CHECK(hipMemcpyAsync(time_series,
                              reinterpret_cast<float *>(&pinned_dm_series[offset]),
                              cur_nsamps * sizeof(float),
                              hipMemcpyHostToDevice, stream));
        break;
      default:
        break;
        // return HD_INVALID_NBITS;
      }
      CHECK(hipStreamSynchronize(stream));

      // 后续处理：基线移除、归一化、匹配滤波和巨脉冲检测
      hd_size nsamps_smooth = hd_size(pl->params.baseline_length / (2 * cur_dt));
      checkThrust("thrust::transform 616\n");
#pragma omp critical
      error = baseline_remover.exec(time_series, cur_nsamps, nsamps_smooth);
      checkThrust("thrust::transform 616\n");
      if (pl->params.verbosity >= 2)
        printf("baseline_remover\n");
      // if (error != HD_NO_ERROR)
      // {
      //   return throw_error(error);
      // }
      checkThrust("thrust::transform 625\n");
#pragma omp critical
      {
        hd_float rms = rms_getter.exec(time_series, cur_nsamps);
        checkThrust("thrust::transform 625\n");
        if (pl->params.verbosity >= 2)
          printf("rms\n");
        checkThrust("thrust::transform 618");
        thrust::transform(thrust::cuda::par.on(stream),
                          pl->d_time_series[tid].begin(), pl->d_time_series[tid].end(),
                          thrust::make_constant_iterator(1.0 / rms),
                          pl->d_time_series[tid].begin(),
                          thrust::multiplies<hd_float>());
        checkThrust("thrust::transform 618");
      }
      CHECK(hipStreamSynchronize(stream));

      // 匹配滤波操作

      hd_size rel_boxcar_max = pl->params.boxcar_max / cur_dm_scrunch;
      hd_size max_nsamps_filtered = cur_nsamps + 1 - rel_boxcar_max;
      hd_size cur_filtered_offset = rel_boxcar_max / 2;
      checkThrust("thrust::transform 640\n");

#pragma omp critical
      {

        matched_filter_plan.prep(time_series, cur_nsamps, rel_boxcar_max);
        checkThrust("thrust::transform 640\n");
        checkThrust("thrust::transform 629");

        hd_float *filtered_series = thrust::raw_pointer_cast(pl->d_filtered_series[tid].data());
        checkThrust("thrust::transform 629");
        if (pl->params.verbosity >= 2)
          printf("matched_filter_plan.prep\n");

        // for (hd_size filter_width = cur_dm_scrunch; filter_width <= pl->params.boxcar_max; filter_width *= 2)
        // {
        //   hd_size rel_filter_width = filter_width / cur_dm_scrunch;
        //   hd_size rel_tscrunch_width = std::max(2 * rel_filter_width / std::max(pl->params.min_tscrunch_width / cur_dm_scrunch, hd_size(1)), hd_size(1));

        //   matched_filter_plan.exec(filtered_series, rel_filter_width, rel_tscrunch_width);

        //   hd_size cur_nsamps_filtered = (max_nsamps_filtered - 1) / rel_tscrunch_width + 1;
        //   thrust::transform(thrust::cuda::par.on(stream),
        //                     thrust::device_pointer_cast(filtered_series),
        //                     thrust::device_pointer_cast(filtered_series + cur_nsamps_filtered),
        //                     thrust::make_constant_iterator(1.0 / sqrt(static_cast<hd_float>(rel_filter_width))),
        //                     thrust::device_pointer_cast(filtered_series),
        //                     thrust::multiplies<hd_float>());
        // }

        // For each boxcar filter
        // Note: We cannot detect pulse widths < current time resolution
        for (hd_size filter_width = cur_dm_scrunch;
             filter_width <= pl->params.boxcar_max;
             filter_width *= 2)
        {
          hd_size rel_filter_width = filter_width / cur_dm_scrunch;
          hd_size filter_idx = get_filter_index(filter_width);

          // Note: Filter width is relative to the current time resolution
          hd_size rel_min_tscrunch_width = std::max(pl->params.min_tscrunch_width / cur_dm_scrunch,
                                                    hd_size(1));
          hd_size rel_tscrunch_width = std::max(2 * rel_filter_width / rel_min_tscrunch_width,
                                                hd_size(1));
          // Filter width relative to cur_dm_scrunch AND tscrunch
          hd_size rel_rel_filter_width = rel_filter_width / rel_tscrunch_width;

          // start_timer(filter_timer);
          checkThrust("thrust::transform 677");

          error = matched_filter_plan.exec(filtered_series, rel_filter_width, rel_tscrunch_width);

          if (pl->params.verbosity >= 2)
            printf("matched_filter_plan.exec\n");
          checkThrust("thrust::transform 677");

          // if (error != HD_NO_ERROR)
          // {
          //   return throw_error(error);
          // }
          // Divide and round up
          hd_size cur_nsamps_filtered = ((max_nsamps_filtered - 1) / rel_tscrunch_width + 1);
          hd_size cur_scrunch = cur_dm_scrunch * rel_tscrunch_width;

          if (pl->params.boxcar_renorm)
          {
            // recompute then RMS of the filtered time series, then use that for rescaling.
            // Note that this method reduces the S/N of injected pulses. For more information
            // see https://ui.adsabs.harvard.edu/abs/2021MNRAS.501.2316G/abstract [Appendix A]

            hd_float rms = rms_getter.exec(filtered_series, cur_nsamps_filtered);
            checkThrust("thrust::transform 691");
            thrust::transform(thrust::device_ptr<hd_float>(filtered_series),
                              thrust::device_ptr<hd_float>(filtered_series) + cur_nsamps_filtered,
                              thrust::make_constant_iterator(hd_float(1.0) / rms),
                              thrust::device_ptr<hd_float>(filtered_series),
                              thrust::multiplies<hd_float>());
            checkThrust("thrust::transform 691");
          }
          else
          {
            // rescale the filtered time series (RMS ~ sqrt(time))
            checkThrust("thrust::transform 702");
            thrust::constant_iterator<hd_float>
                norm_val_iter(1.0 / sqrt((hd_float)rel_filter_width));
            checkThrust("thrust::transform 702");
            checkThrust("thrust::transform 706");
            thrust::transform(thrust::device_ptr<hd_float>(filtered_series),
                              thrust::device_ptr<hd_float>(filtered_series) + cur_nsamps_filtered,
                              norm_val_iter,
                              thrust::device_ptr<hd_float>(filtered_series),
                              thrust::multiplies<hd_float>());
            checkThrust("thrust::transform 706");
          }

          // stop_timer(filter_timer);
          hd_size prev_giant_count = d_giant_peaks_t[tid].size();

          // start_timer(giants_timer);
          checkThrust("thrust::transform 727");
          // #pragma omp critical
          error = giant_finder.exec(filtered_series, cur_nsamps_filtered,
                                    pl->params.detect_thresh,
                                    // pl->params.cand_sep_time,
                                    //  Note: This was MB's recommendation
                                    pl->params.cand_sep_time * rel_rel_filter_width,
                                    d_giant_peaks_t[tid],
                                    d_giant_inds_t[tid],
                                    d_giant_begins_t[tid],
                                    d_giant_ends_t[tid]);

          checkThrust("thrust::transform 727");
          if (pl->params.verbosity >= 2)
            printf("giant_finder.exec\n");
          //           thrust::host_vector<hd_float> h_giant_inds = d_giant_inds_t[tid];  // 将设备向量拷贝到主机
          // for (int i = 0; i < h_giant_inds.size(); ++i) {
          //     std::cout << "h_giant_inds[" << i << "] = " << h_giant_inds[i] << std::endl;
          // }

          // if (error != HD_NO_ERROR)
          // {
          //   return throw_error(error);
          // }

          hd_size rel_cur_filtered_offset = (cur_filtered_offset /
                                             rel_tscrunch_width);

          using namespace thrust::placeholders;
          checkThrust("thrust::transform 739");
          thrust::transform(thrust::cuda::par.on(stream), d_giant_inds_t[tid].begin() + prev_giant_count,
                            d_giant_inds_t[tid].end(),
                            d_giant_inds_t[tid].begin() + prev_giant_count,
                            /*first_idx +*/ (_1 + rel_cur_filtered_offset) * cur_scrunch);

          thrust::transform(thrust::cuda::par.on(stream), d_giant_begins_t[tid].begin() + prev_giant_count,
                            d_giant_begins_t[tid].end(),
                            d_giant_begins_t[tid].begin() + prev_giant_count,
                            /*first_idx +*/ (_1 + rel_cur_filtered_offset) * cur_scrunch);
          thrust::transform(thrust::cuda::par.on(stream), d_giant_ends_t[tid].begin() + prev_giant_count,
                            d_giant_ends_t[tid].end(),
                            d_giant_ends_t[tid].begin() + prev_giant_count,
                            /*first_idx +*/ (_1 + rel_cur_filtered_offset) * cur_scrunch);
          CHECK(hipStreamSynchronize(stream));
          checkThrust("thrust::transform 739");
          // #pragma omp critical
          // {
          d_giant_filter_inds_t[tid].resize(d_giant_peaks_t[tid].size(), filter_idx);
          d_giant_dm_inds_t[tid].resize(d_giant_peaks_t[tid].size(), dm_idx);
          // Note: This could be used to track total member samples if desired
          d_giant_members_t[tid].resize(d_giant_peaks_t[tid].size(), 1);
          checkThrust("thrust::transform 739");
          // }

          stop_timer(giants_timer);
        } // End of filter width loop
      }
      // printf("end of filter loop\n");
    } // DMs for each thread
    CHECK(hipStreamSynchronize(stream));
  }
#pragma omp barrier
  printf("thread complete!\n");
  for (size_t i = 0; i < N_threads; ++i){
    cout<<"thread "<<i<<" grint indx size is:"<<d_giant_inds_t[i].size()<<endl;
  }

  // size_t total_size = 0;
  // for (size_t i = 0; i < N_threads; ++i)
  // {
  //   total_size += d_giant_peaks_t[i].size();
  // }
  thrust::device_vector<hd_float> d_giant_peaks;
  thrust::device_vector<hd_size> d_giant_inds;
  thrust::device_vector<hd_size> d_giant_begins;
  thrust::device_vector<hd_size> d_giant_ends;
   thrust::device_vector<hd_size> d_giant_filter_inds;
   thrust::device_vector<hd_size> d_giant_dm_inds;
   thrust::device_vector<hd_size> d_giant_members;
  size_t offset = 0; // 用于追踪拷贝到新 vector 中的位置

  // 合并 d_giant_peaks
  for (size_t i = 0; i < N_threads; ++i)
  {
    d_giant_peaks.insert(d_giant_peaks.end(), d_giant_peaks_t[i].begin(), d_giant_peaks_t[i].end());
    d_giant_filter_inds.insert(d_giant_filter_inds.end(), d_giant_filter_inds_t[i].begin(), d_giant_filter_inds_t[i].end());
    d_giant_dm_inds.insert(d_giant_dm_inds.end(), d_giant_dm_inds_t[i].begin(), d_giant_dm_inds_t[i].end());
    d_giant_members.insert(d_giant_members.end(), d_giant_members_t[i].begin(), d_giant_members_t[i].end());
    // size_t current_size = d_giant_peaks_t[i].size();
    // thrust::copy(d_giant_peaks_t[i].begin(), d_giant_peaks_t[i].end(), thrust::back_inserter(d_giant_peaks));
  }
  cout<<"d_giant_peaks size is:"<<d_giant_peaks.size()<<endl;

  // 合并 d_giant_inds
  for (size_t i = 0; i < N_threads; ++i)
  {
    d_giant_inds.insert(d_giant_inds.end(), d_giant_inds_t[i].begin(), d_giant_inds_t[i].end());
    // size_t current_size = d_giant_inds_t[i].size();
    // thrust::copy(d_giant_inds_t[i].begin(), d_giant_inds_t[i].end(), thrust::back_inserter(d_giant_inds));
  }
  cout<<"d_giant_inds size is:"<<d_giant_inds.size()<<endl;

  // 合并 d_giant_begins
  for (size_t i = 0; i < N_threads; ++i)
  {
    // size_t current_size = d_giant_begins_t[i].size();
    d_giant_begins.insert(d_giant_begins.end(), d_giant_begins_t[i].begin(), d_giant_begins_t[i].end());
    // thrust::copy(d_giant_begins_t[i].begin(), d_giant_begins_t[i].end(), thrust::back_inserter(d_giant_begins));
  }
  cout<<"d_giant_begins size is:"<<d_giant_begins.size()<<endl;

  // 合并 d_giant_ends
  for (size_t i = 0; i < N_threads; ++i)
  {
    d_giant_ends.insert(d_giant_ends.end(), d_giant_ends_t[i].begin(), d_giant_ends_t[i].end());
    // size_t current_size = d_giant_ends_t[i].size();
    // thrust::copy(d_giant_ends_t[i].begin(), d_giant_ends_t[i].end(), thrust::back_inserter(d_giant_ends));
  }
  cout<<"d_giant_ends size is:"<<d_giant_ends.size()<<endl;

  // 清理 CUDA 流和锁页内存
  for (auto &stream : streams)
  {
    CHECK(hipStreamDestroy(stream));
  }
  CHECK(hipHostFree(pinned_dm_series));
  // 释放内存
  // hipFree(d_time_series);
  // hipFree(d_filtered_series);
  hd_size giant_count = d_giant_peaks.size();
  if (pl->params.verbosity >= 2)
  {
    cout << "Giant count = " << giant_count << endl;
  }

  start_timer(candidates_timer);

  thrust::host_vector<hd_float> h_group_peaks;
  thrust::host_vector<hd_size> h_group_inds;
  thrust::host_vector<hd_size> h_group_begins;
  thrust::host_vector<hd_size> h_group_ends;
  thrust::host_vector<hd_size> h_group_filter_inds;
  thrust::host_vector<hd_size> h_group_dm_inds;
  thrust::host_vector<hd_size> h_group_members;
  thrust::host_vector<hd_float> h_group_dms;

  // if (!too_many_giants)
  //{
  thrust::device_vector<hd_size> d_giant_labels(giant_count);
  hd_size *d_giant_labels_ptr = thrust::raw_pointer_cast(&d_giant_labels[0]);

  RawCandidates d_giants;
  d_giants.peaks = thrust::raw_pointer_cast(&d_giant_peaks[0]);
  d_giants.inds = thrust::raw_pointer_cast(&d_giant_inds[0]);
  d_giants.begins = thrust::raw_pointer_cast(&d_giant_begins[0]);
  d_giants.ends = thrust::raw_pointer_cast(&d_giant_ends[0]);
  d_giants.filter_inds = thrust::raw_pointer_cast(&d_giant_filter_inds[0]);
  d_giants.dm_inds = thrust::raw_pointer_cast(&d_giant_dm_inds[0]);
  d_giants.members = thrust::raw_pointer_cast(&d_giant_members[0]);

  hd_size filter_count = get_filter_index(pl->params.boxcar_max) + 1;

  if (pl->params.verbosity >= 2)
  {
    cout << "Grouping coincident candidates..." << endl;
  }

  ConstRawCandidates *const_d_giants = (ConstRawCandidates *)&d_giants;

  hd_size label_count;
   checkThrust("label_candidate_cluster");
  error = label_candidate_clusters(giant_count,
                                   *const_d_giants,
                                   pl->params.cand_sep_time,
                                   pl->params.cand_sep_filter,
                                   pl->params.cand_sep_dm,
                                   d_giant_labels_ptr,
                                   &label_count);
  checkThrust("label_candidate_cluster");

  if (error != HD_NO_ERROR)
  {
    return throw_error(error);
  }

  hd_size group_count = label_count;
  if (pl->params.verbosity >= 2)
  {
    cout << "Candidate count = " << group_count << endl;
  }

  thrust::device_vector<hd_float> d_group_peaks(group_count);
  thrust::device_vector<hd_size> d_group_inds(group_count);
  thrust::device_vector<hd_size> d_group_begins(group_count);
  thrust::device_vector<hd_size> d_group_ends(group_count);
  thrust::device_vector<hd_size> d_group_filter_inds(group_count);
  thrust::device_vector<hd_size> d_group_dm_inds(group_count);
  thrust::device_vector<hd_size> d_group_members(group_count);

  thrust::device_vector<hd_float> d_group_dms(group_count);

  RawCandidates d_groups;
  d_groups.peaks = thrust::raw_pointer_cast(&d_group_peaks[0]);
  d_groups.inds = thrust::raw_pointer_cast(&d_group_inds[0]);
  d_groups.begins = thrust::raw_pointer_cast(&d_group_begins[0]);
  d_groups.ends = thrust::raw_pointer_cast(&d_group_ends[0]);
  d_groups.filter_inds = thrust::raw_pointer_cast(&d_group_filter_inds[0]);
  d_groups.dm_inds = thrust::raw_pointer_cast(&d_group_dm_inds[0]);
  d_groups.members = thrust::raw_pointer_cast(&d_group_members[0]);

  merge_candidates(giant_count,
                   d_giant_labels_ptr,
                   *const_d_giants,
                   d_groups);

  // Look up the actual DM of each group
  thrust::device_vector<hd_float> d_dm_list(dm_list, dm_list + dm_count);
  thrust::gather(d_group_dm_inds.begin(), d_group_dm_inds.end(),
                 d_dm_list.begin(),
                 d_group_dms.begin());

  // Device to host transfer of candidates
  h_group_peaks = d_group_peaks;
  h_group_inds = d_group_inds;
  h_group_begins = d_group_begins;
  h_group_ends = d_group_ends;
  h_group_filter_inds = d_group_filter_inds;
  h_group_dm_inds = d_group_dm_inds;
  h_group_members = d_group_members;
  h_group_dms = d_group_dms;
  // h_group_flags = d_group_flags;
  //}

  if (pl->params.verbosity >= 2)
  {
    cout << "Writing output candidates, utc_start=" << pl->params.utc_start << endl;
  }

  char buffer[64];
  time_t now = pl->params.utc_start + (time_t)(first_idx / pl->params.spectra_per_second);
  strftime(buffer, 64, HD_TIMESTR, (struct tm *)gmtime(&now));

  std::stringstream ss;
  ss << std::setw(2) << std::setfill('0') << pl->params.beam + 1;

  std::ostringstream oss;

  if (pl->params.coincidencer_host != NULL && pl->params.coincidencer_port != -1)
  {
    try
    {
      ClientSocket client_socket(pl->params.coincidencer_host, pl->params.coincidencer_port);

      strftime(buffer, 64, HD_TIMESTR, (struct tm *)gmtime(&(pl->params.utc_start)));

      oss << buffer << " ";

      time_t now = pl->params.utc_start + (time_t)(first_idx / pl->params.spectra_per_second);
      strftime(buffer, 64, HD_TIMESTR, (struct tm *)gmtime(&now));
      oss << buffer << " ";

      oss << first_idx << " ";
      oss << ss.str() << " ";
      oss << h_group_peaks.size() << endl;
      client_socket << oss.str();
      oss.flush();
      oss.str("");

      for (hd_size i = 0; i < h_group_peaks.size(); ++i)
      {
        hd_size samp_idx = first_idx + h_group_inds[i];
        oss << h_group_peaks[i] << "\t"
            << samp_idx << "\t"
            << samp_idx * pl->params.dt << "\t"
            << h_group_filter_inds[i] << "\t"
            << h_group_dm_inds[i] << "\t"
            << h_group_dms[i] << "\t"
            << h_group_members[i] << "\t"
            << first_idx + h_group_begins[i] << "\t"
            << first_idx + h_group_ends[i] << endl;

        client_socket << oss.str();
        oss.flush();
        oss.str("");
      }
      // client_socket should close when it goes out of scope...
    }
    catch (SocketException &e)
    {
      std::cerr << "SocketException was caught:" << e.description() << "\n";
    }
  }
  else
  {
    if (pl->params.verbosity >= 2)
      cout << "Output timestamp: " << buffer << endl;

    std::string filename = std::string(pl->params.output_dir) + "/" + std::string(buffer) + "_" + ss.str() + ".cand";

    if (pl->params.verbosity >= 2)
      cout << "Output filename: " << filename << endl;

    std::ofstream cand_file(filename.c_str(), std::ios::out);
    if (pl->params.verbosity >= 2)
      cout << "Dumping " << h_group_peaks.size() << " candidates to " << filename << endl;

    if (cand_file.good())
    {
      for (hd_size i = 0; i < h_group_peaks.size(); ++i)
      {
        hd_size samp_idx = first_idx + h_group_inds[i];
        cand_file << h_group_peaks[i] << "\t"
                  << samp_idx << "\t"
                  << samp_idx * pl->params.dt << "\t"
                  << h_group_filter_inds[i] << "\t"
                  << h_group_dm_inds[i] << "\t"
                  << h_group_dms[i] << "\t"
                  << h_group_members[i] << "\t"
                  << first_idx + h_group_begins[i] << "\t"
                  << first_idx + h_group_ends[i] << "\t"
                  << "\n";
      }
    }
    else
      cout << "Skipping dump due to bad file open on " << filename << endl;
    cand_file.close();
  }

  stop_timer(candidates_timer);

  stop_timer(total_timer);

#ifdef HD_BENCHMARK
  if (pl->params.verbosity >= 1)
  {
    cout << "Mem alloc time:          " << memory_timer.getTime() << endl;
    cout << "0-DM cleaning time:      " << clean_timer.getTime() << endl;
    cout << "Dedispersion time:       " << dedisp_timer.getTime() << endl;
    cout << "Copy time:               " << copy_timer.getTime() << endl;
    cout << "Baselining time:         " << baseline_timer.getTime() << endl;
    cout << "Normalisation time:      " << normalise_timer.getTime() << endl;
    cout << "Filtering time:          " << filter_timer.getTime() << endl;
    cout << "Find giants time:        " << giants_timer.getTime() << endl;
    cout << "Process candidates time: " << candidates_timer.getTime() << endl;
    cout << "Total time:              " << total_timer.getTime() << endl;
  }

  hd_float time_sum = (memory_timer.getTime() +
                       clean_timer.getTime() +
                       dedisp_timer.getTime() +
                       copy_timer.getTime() +
                       baseline_timer.getTime() +
                       normalise_timer.getTime() +
                       filter_timer.getTime() +
                       giants_timer.getTime() +
                       candidates_timer.getTime());
  hd_float misc_time = total_timer.getTime() - time_sum;

  /*
  std::ofstream timing_file("timing.dat", std::ios::app);
  timing_file << total_timer.getTime() << "\t"
              << misc_time << "\t"
              << memory_timer.getTime() << "\t"
              << clean_timer.getTime() << "\t"
              << dedisp_timer.getTime() << "\t"
              << copy_timer.getTime() << "\t"
              << baseline_timer.getTime() << "\t"
              << normalise_timer.getTime() << "\t"
              << filter_timer.getTime() << "\t"
              << giants_timer.getTime() << "\t"
              << candidates_timer.getTime() << endl;
  timing_file.close();
  */

#endif // HD_BENCHMARK

  if (too_many_giants)
  {
    return HD_TOO_MANY_EVENTS;
  }
  else
  {
    return HD_NO_ERROR;
  }
}

void hd_destroy_pipeline(hd_pipeline pipeline)
{
  if (pipeline->params.verbosity >= 2)
  {
    cout << "\tDeleting pipeline object..." << endl;
  }

  dedisp_destroy_plan(pipeline->dedispersion_plan);

  // Note: This assumes memory owned by pipeline cleans itself up
  if (pipeline)
  {
    delete pipeline;
  }
}
